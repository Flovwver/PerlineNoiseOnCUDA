﻿#include "hip/hip_runtime.h"

#include "cmath"

#include <stdio.h>
#include <fstream>

const unsigned int height = 1024;
const unsigned int width = 1024;

hipError_t addWithCuda(float *c);
bool saveArrayInTxt(float* array);

__device__ float Frac(float xFloat) 
{
    int xInt = xFloat;
    return xFloat - xInt;
}

__device__ float Dot(float2 vectorLeft, float2 vectorRight)
{
    return vectorLeft.x * vectorRight.x + vectorLeft.y * vectorRight.y;
}

__device__ float Rand(float2 x)
{
    float2 randomNumber = make_float2(Frac(sin(Dot(x, make_float2(78.233, 12.9898))) * 43758.5453), Frac(sin(Dot(x, make_float2(78.233 * 2, 12.9898 * 2))) * 43758.5453));
    return abs(randomNumber.x + randomNumber.y) * 0.5;
}

__device__ float Rand(int2 xInt)
{
    float2 xFloat;
    xFloat.x = xInt.x;
    xFloat.y = xInt.y;
    Rand(xFloat);
}

__device__ float Noise(int2 uv)
{
    float randomNumber = Rand(uv);
    return randomNumber;
}

__device__ float BilinearInterpolation(float f00, float f01, float f10, float f11, int x0, int x1, int y0, int y1, float2 uv)
{
    float fR1 = (x1 - uv.x) / (x1 - x0) * f00 + (uv.x - x0) / (x1 - x0) * f10;
    float fR2 = (x1 - uv.x) / (x1 - x0) * f01 + (uv.x - x0) / (x1 - x0) * f11;
    return (y1 - uv.y) / (y1 - y0) * fR1 + (uv.y - y0) / (y1 - y0) * fR2;
}

__device__ float ShellInterpolation(float2 uv, int coeficient)
{
    return BilinearInterpolation(Noise(make_int2(uv.x * coeficient, uv.y * coeficient)), Noise(make_int2(uv.x * coeficient, uv.y * coeficient+1)),
        Noise(make_int2(uv.x * coeficient + 1, uv.y * coeficient)), Noise(make_int2(uv.x * coeficient+1, uv.y * coeficient+1)), uv.x * coeficient,
        uv.x * coeficient + 1, uv.y * coeficient, uv.y * coeficient + 1, make_float2(uv.x * coeficient, uv.y * coeficient));
}

__device__ float PerlinNoise(float2 uv)
{
    float color = 0.f;
    int numberOfCycles = 10;
    for (int i = numberOfCycles; i >= 1; i--) 
    {
        color = color * 0.4 + ShellInterpolation(uv, pow(3, i));
    }
    return color * 0.8;
}

__global__ void addKernel(float *c)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    c[i* width + j] = Frac(PerlinNoise(make_float2((float) i / height, (float) j / width)));
}

int main()
{
    float* c = new float[height * width];

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    saveArrayInTxt(c);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c)
{
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, height * width * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<height, width>>>(dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, height * width * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    
    return cudaStatus;
}

bool saveArrayInTxt(float* array) 
{
    FILE* file;
    if ((file = fopen("perlineNoise.bin", "w")) == NULL) {
        printf("error\n");
        return false;
    }
    else {
        fwrite(array, sizeof(float), height * width, file);
    }
    fclose(file);
    printf("Successful \n");
    return true;
}
